#include "hip/hip_runtime.h"
#include "CUDAFunc.cuh"

// Conv, Pool layer
float *dev_Conv_kernel[nConvLayer], *dev_Conv_grad[nConvLayer], *dev_Conv_m_prev[nConvLayer], *dev_Conv_v_prev[nConvLayer];
char *devicePoolMark[nPoolLayer];
// FC Layer
float *dev_FC_w[nFCLayer], *dev_FC_grad[nFCLayer], *dev_FC_m_prev[nFCLayer], *dev_FC_v_prev[nFCLayer];
// shared node
float *dev_Node[nCnPLayer + nFCLayer + 1], *dev_Node_delta[nCnPLayer + nFCLayer];

// test drop node
float *dev_drop[nFCLayer - 1];

__global__ void forward_layer(float *d_weights, int weightOffset, int weightsPerNeuron, float *d_ins, int neuronsPrev, float *d_outs, bool softmax)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int wid = threadIdx.x * weightsPerNeuron + (blockIdx.x * weightsPerNeuron);
	
	float a = .0f;

	for (int i = 0; i < weightsPerNeuron; ++i){
		a += d_weights[wid + i] * d_ins[i];
				//printf("\n [tid:%d], d_weights[%d](%.1f) * d_ins[%d](%.1f) = %.1f", tid, wid+i, d_weights[wid+i], i, d_ins[i]);
		//printf("\n [tid:%d], d_weights[%d](%f) * d_ins[%d](%f) = %f", tid, wid + i, d_weights[wid + i], i, d_ins[i], a);
		//printf("d_outs[%d] : %f\n", tid, a);
	}


	if (softmax) d_outs[tid] = a;
	else		d_outs[tid] = (a > 0.0f ? a : a*0.01f);
}

__global__ void CUDA_Conv2D(float *I, float* M, float *P, int inmap, int outmap, int width, int height, int kernel_size, int padding)
{
	//CUDA_Conv2D << < Conv->Outmap, Conv->Output_data->y * Conv->Output_data->z >> > (dev_Node[layer_idx], dev_Conv_kernel[kernel_num], dev_Node[layer_idx + 1],
	//Conv->Inmap, Conv->Outmap, Conv->Input_data->y, Conv->Input_data->z, Conv->kernel->y, Padding);

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = 0, tidx = 0, mask_idx = 0, oddCnt = 0;

	int x, y, outputSize, outputDim, _padding = 0;
	int mask_dim = kernel_size*kernel_size;
	float val = 0.0;

	if (padding == 0){
		outputSize = (width - (kernel_size - 1)) * (width - (kernel_size - 1));
		outputDim = width - (kernel_size - 1);
	}
	else{
		outputSize = width * height;
		outputDim = width;
		_padding = kernel_size / 2;
	}

	for (int mapCnt = 0; mapCnt < inmap; mapCnt++, tidx += (width * (outputDim))){
		//printf("\n mapCnt : %d", mapCnt);
		idx = tidx + threadIdx.x;

		x = idx / outputDim;
		y = idx % outputDim;

		oddCnt = mapCnt * mask_dim;

		for (int i = 0; i < kernel_size; i++){
			int xx = x + i - _padding;
			for (int j = 0; j < kernel_size; j++){
				int yy = y + j - _padding;

				// ���߿� ���ǹ� ������ ��
				if (padding == 0){
					val += I[xx * width + yy] * M[((tid / (outputSize)) * mask_dim * inmap + oddCnt) + (i * kernel_size + j)];
					//printf("\n mapCnt : %d, idx : %d, tid : %d, val += I[%d] * M[%d] : %.4f * %.4f = %.3f , i : %d , j : %d, tidx : %d, width : %d, outputDim : %d",
						//mapCnt, idx, tid, xx * width + yy, ((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j),
						//I[xx * width + yy], M[((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j)], val, i, j, tidx,width, outputDim);
					//printf("\n blockIdx : %d, threadIdx : %d, tidx : %d , blockDimx : %d, width : %d, outputDim : %d", tid, idx, blockIdx.x, threadIdx.x, tidx, blockDim.x, width, outputDim);
				}
				else{
					if ((xx >= 0 && yy >= 0) &&
						((xx < (width*(mapCnt + 1))) &&
						(yy < height)) &&
						((xx * width + yy) < (outputSize * (mapCnt + 1))) &&
						((xx * width + yy) >= tidx)){

						val += I[xx * width + yy] * M[((tid / (outputSize)) * mask_dim * inmap + oddCnt) + (i * kernel_size + j)]; 
						/*
						printf("\n mapCnt : %d, idx : %d, tid : %d, val += I[%d] * M[%d] : %.4f * %.4f = %.3f , i : %d , j : %d, tidx : %d, width : %d, outputDim : %d",
						mapCnt, idx, tid, xx * width + yy, ((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j),
							//I[xx * width + yy], M[((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j)], val, i, j, tidx, width, outputDim);
							*/
						//printf("\n mapCnt : %d, idx : %d, tid : %d, val += I[%d] * M[%d] : %.4f * %.4f = %.3f x : %d, y : %d, outputDim : %d",
							//mapCnt, idx, tid, xx * width + yy, ((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j),
							//I[xx * width + yy], M[((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j)], val, x, y, outputDim);
						/*
						if (tid == 0){
							printf("\n mapCnt : %d, idx : %d, tid : %d, val += I[%d] * M[%d] : %.4f * %.4f = %.3f x : %d, y : %d",
								mapCnt, idx, tid, xx * width + yy, ((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j),
								I[xx * width + yy], M[((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j)], val, x, y);
						}
						*/
					}
				}
				/*
				printf("\n mapCnt : %d, idx : %d, tid : %d, val += I[%d] * M[%d] : %.4f * %.4f = %.3f , i : %d , j : %d, tidx : %d, width : %d, outputDim : %d",
					mapCnt, idx, tid, xx * width + yy, ((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j),
					I[xx * width + yy], M[((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j)], val, i, j, tidx, width, outputDim);
					*/
				if (tid == 0){
					//printf("\n mapCnt : %d, idx : %d, tid : %d, val += I[%d] * M[%d] : %.4f * %.4f = %.3f ",
						//mapCnt, idx, tid, xx * width + yy, ((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j),
						//I[xx * width + yy], M[((tid / (outputSize))*mask_dim*inmap + oddCnt) + (i * kernel_size + j)], val);
				}
			}
		}
		P[tid] = (val > 0 ? val : val * 0.01);	// Act. func.

		//printf("\n P[%d] : %.3f", tid, P[tid]);
		//if (tid == 0)	printf("\n P[%d] : %.3f", tid, P[tid]);
	}
}

__global__ void CUDA_MaxPooling(float *I, float *P, char *pool_mark, int inmap, int width, int height){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = 0, tidx = 0, cnt = 0;
	char pool_idx = 0;

	const int outputDim = width / 2;

	//float _max = 0;

	int init_h = (threadIdx.x / outputDim) * 2;
	int init_w = (threadIdx.x % outputDim) * 2;

	float _max = I[(blockIdx.x * width * width) + (init_h * width + init_w)];

	//printf("\n B : %d, T : %d, tid  :%d, Max : %.4f", blockIdx.x, threadIdx.x, tid, _max);
	//const int outputDim = width / 2;

	for (int h = 0; h < 2; h++){

		int _h = h + (threadIdx.x / outputDim) * 2;

		for (int w = 0; w < 2; w++){

			int _w = w + (threadIdx.x % outputDim) * 2;  // ����

			if (I[(blockIdx.x * width * width) + (_h * width + _w)] > _max){
				_max = I[(blockIdx.x * width * width) + (_h * width + _w)];
				pool_idx = cnt;
			}
			cnt++;
		}
	}
	P[tid] = _max;
	pool_mark[tid] = pool_idx;

	//printf("\n B:%d,T:%d, p[%d] : %.2f , pool_mark[%d] : %d", 
	//	blockIdx.x, threadIdx.x, tid, P[tid], tid, pool_mark[tid]);
}

////////////////////////////////////////////////////////////

void CUDA_CnP_MemAlloc(ConvLayer *Conv, PoolLayer *Pool){

	for (int i = 0; i < nConvLayer; i++){
		hipMalloc((void **)&dev_Conv_kernel[i], (Conv[i].Inmap * Conv[i].Outmap) * (Conv[i].kernel->y * Conv[i].kernel->z) * sizeof(float));
		hipMalloc((void **)&dev_Conv_grad[i], (Conv[i].Inmap * Conv[i].Outmap) * (Conv[i].kernel->y * Conv[i].kernel->z) * sizeof(float));
		hipMalloc((void **)&dev_Conv_m_prev[i], (Conv[i].Inmap * Conv[i].Outmap) * (Conv[i].kernel->y * Conv[i].kernel->z) * sizeof(float));
		hipMalloc((void **)&dev_Conv_v_prev[i], (Conv[i].Inmap * Conv[i].Outmap) * (Conv[i].kernel->y * Conv[i].kernel->z) * sizeof(float));

		reshape3Dto1D(Conv[i].kernel);

		hipMemcpy(dev_Conv_kernel[i], Conv[i].kernel->mem1D,
			(Conv[i].Inmap * Conv[i].Outmap) * (Conv[i].kernel->y * Conv[i].kernel->z) * sizeof(float), hipMemcpyHostToDevice);
	}
	for (int i = 0; i < nPoolLayer; i++){
		hipMalloc((void **)&devicePoolMark[i], Pool[i].Outmap * Pool[i].pool_mark->y * Pool[i].pool_mark->z * sizeof(char));
	}

}
void CUDA_FC_MemAlloc(FCLayer *FC){

	for (int i = 0; i < nFCLayer; i++){
		if (i < nFCLayer - 1){
			hipMalloc((void**)&dev_FC_w[i], FC[i].Input_size * FC[i + 1].Input_size * sizeof(float));
			hipMalloc((void**)&dev_FC_grad[i], FC[i].Input_size * FC[i + 1].Input_size * sizeof(float));
			hipMalloc((void**)&dev_FC_m_prev[i], FC[i].Input_size * FC[i + 1].Input_size * sizeof(float));
			hipMalloc((void**)&dev_FC_v_prev[i], FC[i].Input_size * FC[i + 1].Input_size * sizeof(float));

			hipMemcpy(dev_FC_w[i], FC[i].Weight->mem1D, FC[i].Input_size * FC[i + 1].Input_size * sizeof(float), hipMemcpyHostToDevice);
		}
		else{
			hipMalloc((void**)&dev_FC_w[i], FC[i].Input_size * FC[i].Output_size * sizeof(float));
			hipMalloc((void**)&dev_FC_grad[i], FC[i].Input_size * FC[i].Output_size* sizeof(float));
			hipMalloc((void**)&dev_FC_m_prev[i], FC[i].Input_size * FC[i].Output_size * sizeof(float));
			hipMalloc((void**)&dev_FC_v_prev[i], FC[i].Input_size * FC[i].Output_size * sizeof(float));

			hipMemcpy(dev_FC_w[i], FC[i].Weight->mem1D, FC[i].Input_size * FC[i].Output_size * sizeof(float), hipMemcpyHostToDevice);
		}
	}

}
void CUDA_ioNode_MemAlloc(MemBlock32F *ioNode){

	// �Է� ���
	hipMalloc((void**)&dev_Node[0], ioNode[0].total * sizeof(float));
	for (int i = 1; i < nCnPLayer + nFCLayer + 1; i++){
		hipMalloc((void**)&dev_Node[i], ioNode[i].total * sizeof(float));
		hipMalloc((void**)&dev_Node_delta[i], ioNode[i].total * sizeof(float));
	}

#if Dropout
	for (int i = 1; i < nFCLayer; i++)
		hipMalloc((void**)&dev_drop[i - 1], (ioNode + nCnPLayer + i)->total * sizeof(int));
#endif

}

void CUDA_FC_Forward(FCLayer *FC, int &layer_idx){

	int block = 0, thread = 0;
	int offset = (FC - 1)->Input_size * (FC - 1)->Output_size;
	bool softmax = false;

	if (layer_idx == 0) offset = 0;
	if (layer_idx == nFCLayer - 1) softmax = true;

	if (FC->Output_size % MAX_THREAD == 0){
		block = FC->Output_size / MAX_THREAD;
	}
	else{
		block = (FC->Output_size / MAX_THREAD) + 1;
	}

	if (block > 1) thread = MAX_THREAD;
	else		   thread = FC->Output_size;

	//printf("forward layer\n");
	//printf("offset : %d, FC->Input_size : %d\n", offset, FC->Input_size);
	//printf("block : %d, thread : %d\n", block, thread);
	//		forward_layer << < 1, FC->Output_size >> > (dev_FC_w[layer_idx], offset, FC->Input_size, dev_Node[nCnPLayer + layer_idx], FC->Input_size, dev_Node[nCnPLayer + layer_idx + 1], softmax);
	forward_layer << < block, thread >> > (dev_FC_w[layer_idx], offset, FC->Input_size, dev_Node[nCnPLayer + layer_idx], FC->Input_size, dev_Node[nCnPLayer + layer_idx + 1], softmax);

	layer_idx++;
}
void CUDA_Conv_Forward(ConvLayer *Conv, int Padding, int &layer_idx, int &kernel_num){
	//CUDA_Conv_Forward(&Conv[0], ConvPad_info[0], CnP_layer_idx, kernel_num);

	CUDA_Conv2D << < Conv->Outmap, Conv->Output_data->y * Conv->Output_data->z >> > (dev_Node[layer_idx], dev_Conv_kernel[kernel_num], dev_Node[layer_idx + 1],
		Conv->Inmap, Conv->Outmap, Conv->Input_data->y, Conv->Input_data->z, Conv->kernel->y, Padding);


	// update�� ���.. update CUDA ���� �� ����
	hipMemcpy(Conv->Output_data->mem1D, dev_Node[layer_idx], Conv->Output_data->total * sizeof(float), hipMemcpyDeviceToHost);
	reshape1Dto3D(Conv->Output_data);

	layer_idx++; // �ε��� ����
	kernel_num++;

}
void CUDA_Pool_Forward(PoolLayer *Pool, int &layer_idx, int &poolMark_num){

	CUDA_MaxPooling << < Pool->Inmap, Pool->Output_data->y * Pool->Output_data->z >> >(dev_Node[layer_idx], dev_Node[layer_idx + 1], devicePoolMark[poolMark_num]
		, Pool->Inmap, Pool->Input_data->y, Pool->Input_data->z);

	hipMemcpy(Pool->Output_data->mem1D, dev_Node[layer_idx + 1], Pool->Output_data->total * sizeof(float), hipMemcpyDeviceToHost);
	reshape1Dto3D(Pool->Output_data);	 // update�� ���.. update CUDA ���� �� ����

	layer_idx++;	// �ε��� ����
	poolMark_num++;
}
