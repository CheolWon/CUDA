//cuda 1d
#include <iostream>
#include <time.h>
#include <math.h>
#include <stdio.h>			
#include <stdlib.h>
#include <string.h>

#include "Define.h"
#include "MemBlock.h"
#include "Config.h"
#include "utils.h"
#include "CNNFunc.h"
#include "CUDAfunc.cuh"
#include "hip/hip_runtime.h"

#define lrn_alpha		0.0001		// 0.0001f
#define lrn_beta		0.5		// 0.75f
#define	lrn_radius		5		// 5
#define lrn_k			2.0		// 1.0f
		
int main(){
	char ch = 0; 
	bool validFlag = false;
	float validAcc = 0;
	srand((unsigned int)time(NULL));
	float		 train_input[Image_W*Image_depth][Image_H], fc_target[NUM_OUTPUTS];
	unsigned int train_label;
#if MNIST
	char path[100] = "E:/ESL-LAB/�����ڷ�/��������/170725_CUDA_MNIST/MNIST_TRAIN/";
#endif
#if CIFAR10
	char path[100] = "E:/ESL-LAB/�����ڷ�/��������/170725_CUDA_MNIST/CIFAR_TRAIN/";
#endif
	float target[NUM_OUTPUTS][NUM_OUTPUTS] = { { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 0, 1, 0, 0, 0, 0, 0, 0, 0, 0 }, { 0, 0, 1, 0, 0, 0, 0, 0, 0, 0 },  // '0','1','2'
											{ 0, 0, 0, 1, 0, 0, 0, 0, 0, 0 }, { 0, 0, 0, 0, 1, 0, 0, 0, 0, 0 }, { 0, 0, 0, 0, 0, 1, 0, 0, 0, 0 },	  // '3','4','5' 
											{ 0, 0, 0, 0, 0, 0, 1, 0, 0, 0 }, { 0, 0, 0, 0, 0, 0, 0, 1, 0, 0 }, { 0, 0, 0, 0, 0, 0, 0, 0, 1, 0 },	  // '6','7','8'											
											{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 1 } }; 																	  // '9'

	///////////////////////////////////////////////////////////////////////////////////
	// ��Ʈ��ũ ���� ����
	int input_size, map[nConvLayer], kernel_size[nConvLayer];
	int Layerinfo[nCnPLayer] = { 1, 0, 1, 0 };		// 1:conv,0:pool
	int ConvPad_info[nConvLayer] = { 0, 0 };		// 1:padding
	//int ConvPad_info[nConvLayer] = { 1, 1 };		// 1:padding
	input_size = Image_H;							// Input data	
	
	map[0] = 8, kernel_size[0] = 5;
	map[1] = 16, kernel_size[1] = 3;  

	char FilePath[30] = "./mnist_0_3";
	
	/*/
	map[0] = 16, kernel_size[0] = 5;
	map[1] = 32, kernel_size[1] = 5;
	*/
	///////////////////////////////////////////////////////////////////////////////////
	ConvLayer	Conv[nConvLayer];
	PoolLayer	Pool[nPoolLayer];
	FCLayer		FC[nFCLayer];
	memset(&Conv, 0, sizeof(ConvLayer[nConvLayer]));
	memset(&Pool, 0, sizeof(PoolLayer[nPoolLayer]));
	memset(&FC, 0, sizeof(FCLayer[nFCLayer]));

	// Layer �� ���� ���
	MemBlock32F ioNode[nCnPLayer + nFCLayer + 1], CNN_Delta[nCnPLayer + nFCLayer], prevPatch[nCnPLayer + 1];
	// Conv,Pool Layer	
	MemBlock32F kernel[nConvLayer], ConvPad[nConvLayer], deltaPad[nConvLayer - 1], CNN_Moment[nConvLayer],
		CNN_gradSum[nConvLayer], CNN_mean_prev[nConvLayer], CNN_var_prev[nConvLayer];
	MemBlock8C	pool_mark[nPoolLayer];
	// FC Layer
	MemBlock32F FC_Weight[nFCLayer], FC_Moment[nFCLayer], FC_gradSum[nFCLayer], FC_mean_prev[nFCLayer], FC_var_prev[nFCLayer];

	setLayer(input_size, Image_depth, map, kernel_size, ConvPad_info, Layerinfo, prevPatch,
		kernel, ConvPad, pool_mark, deltaPad, CNN_Delta, CNN_Moment, CNN_gradSum, CNN_mean_prev, CNN_var_prev, ioNode, Conv, Pool,
		FC_Weight, FC_Moment, FC_gradSum, FC_mean_prev, FC_var_prev, FC);
	//cout << "ifCUDA" << endl;
#if CUDA
	int kernel_num = 0, poolMark_num = 0, CnP_layer_idx = 0, FC_layer_idx = 0;

	// shared node (CUDA memAlloc)
	CUDA_ioNode_MemAlloc(ioNode);

	// FC Layer (CUDA memAlloc)
	const int node1 = FC[0].Input_size, node2 = FC[1].Input_size, node3 = FC[2].Input_size, node4 = NUM_OUTPUTS;

	//printf("FC1.input_size : %d, FC2.input_size : %d, FC3.input_size : %d", FC[0].Input_size, FC[1].Input_size, FC[2].Input_size);
	CUDA_FC_MemAlloc(FC);
	//printf("\n");

	// Conv/Pool Layer (CUDA memAlloc)
	CUDA_CnP_MemAlloc(Conv, Pool);
#endif
	//cout << "ifTraning" << endl;
	///////////////////////////////////////////////////////////////////////////////////
	// Training ...  //////////////////////////////////////////////////////////////////
#if Training
	int num_cnt[NUM_OUTPUTS], _num_cnt[NUM_OUTPUTS], randVal, epoch, num_data;
	float total_loss = 0, loss = 0, acc = 0, _LearningRate = learningRate, batch_loss = 0, time = 0;

	if (Test){ // Test 
		loadWeight(Conv, FC);
		for (int i = 0; i < nFCLayer; i++){
			hipMemcpy(dev_FC_w[i], FC[i].Weight->mem1D, FC[i].Weight->total * sizeof(float), hipMemcpyHostToDevice);
		}
		for (int i = 0; i < nConvLayer; i++){
			reshape3Dto1D(Conv[i].kernel);
			hipMemcpy(dev_Conv_kernel[i], Conv[i].kernel->mem1D, Conv[i].kernel->total * sizeof(float), hipMemcpyHostToDevice);
		}
		epoch = 1;
		num_data = NUM_TEST_DATA;
		for (int i = 0; i < NUM_OUTPUTS; i++)  _num_cnt[i] = _NUM_TEST_DATA;
	}
	else{	  // Train
		epoch = iter;
		num_data = NUM_TRAIN_DATA;
	}

	for (int i = 0; i < 1; ++i) {  // Epoch
		// epoch �� �ʱ�ȭ ����
		for (int i = 0; i < NUM_OUTPUTS; i++)  num_cnt[i] = _NUM_TRAIN_DATA; // num_cnt[i] = MNIST;
		if(!Test)	 for (int i = 0; i < NUM_OUTPUTS; i++)  _num_cnt[i] = _NUM_VALID_DATA;
		total_loss = acc = 0;

		// learning rate decay 
		if ((i + 1) % 10 == 0) _LearningRate *= 0.9;
		//cout << "validation" << endl;

		for (int q = 0; q < 1; q++) {
			randVal = ((rand() % 100) % NUM_OUTPUTS);
				//readDataset(train_input, _num_cnt, train_label, randVal, path, 0); 
			readDataset(train_input, _num_cnt, train_label, randVal, FilePath, 0);
			copyInputData1D(train_input, &ioNode[0]);
			
			for (int i = 0; i < 28; i++) {
				for (int j = 0; j < 28; j++) {
					printf("%1.0f ", Conv[0].Input_data->mem1D[i * 28 + j]);
				}
				printf("\n");
			}
			
			hipMemcpy(dev_Node[0], Conv[0].Input_data->mem1D, Image_H * Image_W * Image_depth * sizeof(float), hipMemcpyHostToDevice);
			//cout << "if0" << endl;
			///// Ư¡����� ///////////////////////////////////////////////////////////

#if 1
			CUDA_Conv_Forward(&Conv[0], ConvPad_info[0], CnP_layer_idx, kernel_num);

			for (int i = 0; i < 10; i++) {
				//printf("%lf ", Conv[0].Output_data->mem1D[i]);
			}
			
			//printf("\n");

			//printf("Conv[0].In : %d\n", Conv[0].Input_data->total);
			//printf("Conv[0].Out : %d\n", Conv[0].Output_data->total);
			CUDA_Pool_Forward(&Pool[0], CnP_layer_idx, poolMark_num);
			//printf("--------------------------------Conv2----------------------------------------\n");
			CUDA_Conv_Forward(&Conv[1], ConvPad_info[1], CnP_layer_idx, kernel_num);

			//printf("Conv[1].In : %d\n", Conv[1].Input_data->total);
			//printf("Conv[1].Out : %d\n", Conv[1].Output_data->total);
			CUDA_Pool_Forward(&Pool[1], CnP_layer_idx, poolMark_num);

			//printf("Pool[1].In : %d\n", Pool[1].Input_data->total);
			//printf("Pool[1].Out : %d\n", Pool[1].Output_data->total);
			/*
			for (int i = 0; i < 400; i++) {
				printf("%lf ", Pool[1].Output_data->mem1D[i]);
			}
			printf("\n");
			*/
#endif
			///// �з��� //////////////////////////////////////////////////////////////

			memcpy(FC[0].Input_data->mem1D, Pool[1].Output_data->mem1D, FC[0].Input_data->total);

			
			for (int i = 0; i < 400; i++) {
				//printf("%lf ", FC[0].Input_data->mem1D[i]);
			}
			

			//printf("\n");
			CUDA_FC_Forward(&FC[0], FC_layer_idx); // ��Ӿƿ� �׽�Ʈ

			//printf("FC[0].In : %d\n", FC[0].Input_data->total);
			//printf("FC[0].Out : %d\n", FC[0].Output_data->total);


			hipMemcpy(FC[0].Output_data->mem1D, dev_Node[nCnPLayer + 1], 200 * sizeof(float), hipMemcpyDeviceToHost);

			for (int i = 0; i < FC[0].Output_data->total; i++) {
				//printf("%lf ", FC[0].Output_data->mem1D[i]);
			}
			//printf("\n");

			//CUDA_FC_Forward_Dropout(&FC[1], FC_layer_idx, validFlag);
			CUDA_FC_Forward(&FC[1], FC_layer_idx);

			//printf("FC[1].In : %d\n", FC[1].Input_data->total);
			//printf("FC[1].Out : %d\n", FC[1].Output_data->total);

			//CUDA_FC_Forward(&FC[2], FC_layer_idx);

			// output (toHost)
			hipMemcpy(FC[1].Output_data->mem1D, dev_Node[nCnPLayer + nFCLayer], node4 * sizeof(float), hipMemcpyDeviceToHost);

			/////////////////////////////////////////////////////////////////////////////
			// Softmax, Output Error, Loss function (cross-entropy)
			int label = train_label;
			memcpy(fc_target, target[label], sizeof(target[label]));

			loss = 0;
			Softmax_CrossEntropy(&acc, &loss, fc_target, &FC[nFCLayer - 1]);
			total_loss += loss;
#endif
		}
		acc /= num_data;
		total_loss /= num_data;

		if (Test)  printf("\n Testset[%d]. Accuracy : %.2f%% ", NUM_TEST_DATA, acc);

	}

	// �޸� ���� //////////////////////////////////////////////////////////////
	// # �������
	for (int i = 0; i < nCnPLayer + nFCLayer + 1; i++) memRelease32F(&ioNode[i]);
	for (int i = 0; i < nCnPLayer + nFCLayer; i++)  memRelease32F(&CNN_Delta[i]);

	// # Ư¡�����
	for (int i = 0; i < nConvLayer; i++){
		memRelease32F(&kernel[i]);
		memRelease32F(&CNN_Moment[i]);
		memRelease32F(&CNN_gradSum[i]);
		if (i < nConvLayer - 1) memRelease32F(&deltaPad[i]);
	}
	for (int i = 0; i < nPoolLayer; i++)		memRelease8C(&pool_mark[i]);

	// # �з���
	for (int i = 0; i < nFCLayer; i++){
		memRelease32F(&FC_Weight[i]);
		memRelease32F(&FC_Moment[i]);
		memRelease32F(&FC_gradSum[i]);
	}

#if CUDA
	// Shared node
	for (int i = 0; i < nCnPLayer + nFCLayer + 1; i++){
		hipFree(dev_Node[i]);
		hipFree(dev_Node_delta[i]);
	}
	// Conv layer
	for (int i = 0; i < nConvLayer; i++){
		hipFree(dev_Conv_kernel[i]);
		hipFree(dev_Conv_grad[i]);
		hipFree(dev_Conv_m_prev[i]);
		hipFree(dev_Conv_v_prev[i]);
	}
	// Pool layer
	for (int i = 0; i < nPoolLayer; i++){
		hipFree(devicePoolMark[i]);
	}
	// FC layer
	for (int i = 0; i < nFCLayer; i++){
		hipFree(dev_FC_w[i]);
		hipFree(dev_FC_grad[i]);
		hipFree(dev_FC_m_prev[i]);
		hipFree(dev_FC_v_prev[i]);
	}
#endif

	return 0;
}

