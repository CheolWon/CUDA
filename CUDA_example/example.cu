#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

#define ITER 65536

// CPU version of the vector add function
void vector_add_cpu(int *a, int *b, int *c, int n) {
    int i;

    // Add the vector elements a and b to the vector c
    for (i = 0; i < n; ++i) {
    c[i] = a[i] + b[i];
    }
}

// GPU version of the vector add function
__global__ void vector_add_gpu(int *gpu_a, int *gpu_b, int *gpu_c, int n) {
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = threadIdx.x;
    // No for loop needed because the CUDA runtime
    // will thread this ITER times
    gpu_c[idx] = gpu_a[idx] * gpu_b[idx];
}

int main() {

    int *a, *b, *c, *gpu_r;
    int *gpu_a, *gpu_b, *gpu_c;

    a = (int *)malloc(ITER * sizeof(int));
    b = (int *)malloc(ITER * sizeof(int));
    c = (int *)malloc(ITER * sizeof(int));
    gpu_r = (int *)malloc(ITER * sizeof(int)); 

    // We need variables accessible to the GPU,
    // so cudaMallocManaged provides these
    hipMalloc((void**)&gpu_a, ITER * sizeof(int));
    hipMalloc((void**)&gpu_b, ITER * sizeof(int));
    hipMalloc((void**)&gpu_c, ITER * sizeof(int));

    for (int i = 0; i < ITER; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = i;
	gpu_r[i] = i;
    }

    // Call the CPU function and time it
    auto cpu_start = Clock::now();
    vector_add_cpu(a, b, c, ITER);
    auto cpu_end = Clock::now();
    std::cout << "vector_add_cpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(cpu_end - cpu_start).count()
    << " nanoseconds.\n";

    for(int i=0;i<10;i++)
	std::cout << "vector_add_cpu : " << c[i] << " ";
    std::cout<<"\n";
    /*
    for(int i=0;i<10;i++)
	std::cout << "result : " << result[i] << " ";
    std::cout<<"\n";
    */

    // Call the GPU function and time it
    // The triple angle brakets is a CUDA runtime extension that allows
    // parameters of a CUDA kernel call to be passed.
    // In this example, we are passing one thread block with ITER threads.
    //cudaMemcpy(void* dst, const void* src, size_t count, cudaMemcpyHostToDevice/cudaMemcpyDeviceToHost);
    hipMemcpy(gpu_a, a, ITER * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, ITER * sizeof(int), hipMemcpyHostToDevice);
    
    auto gpu_start = Clock::now();
    //vector_add_gpu <<<2, ITER/2>>> (gpu_a, gpu_b, gpu_c, ITER);
    vector_add_gpu <<<1, ITER>>> (gpu_a, gpu_b, gpu_c, ITER);
    hipDeviceSynchronize();
    auto gpu_end = Clock::now();
    std::cout << "vector_add_gpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_end - gpu_start).count()
    << " nanoseconds.\n";
     
    /*
    for(int i=0;i<10;i++)
	std::cout << "vector_add_gpu : " << gpu_r[i] << " ";
    std::cout<<"\n";
    */

    hipMemcpy(gpu_r, gpu_c, ITER * sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout<<"result of gpu_c"<<std::endl;
    for(int i=0;i<10;i++)
	std::cout << "vector_add_gpu : " << gpu_r[i] << " ";
    std::cout<<"\n";

    //Free the GPU-function based memory allocations
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // Free the CPU-function based memory allocations
    free(a);
    free(b);
    free(c);
    free(gpu_r);
    /* 
    int InputData[5] = {1, 2, 3, 4, 5};
    int OutputData[5] = {0};
 
    int* GraphicsCard_memory;
 
    //그래픽카드 메모리의 할당
    cudaMalloc((void**)&GraphicsCard_memory, 5*sizeof(int));
 
    //PC에서 그래픽 카드로 데이터 복사
    cudaMemcpy(GraphicsCard_memory, InputData, 5*sizeof(int), cudaMemcpyHostToDevice);
 
    //그래픽 카드에서 PC로 데이터 복사
    cudaMemcpy(OutputData, GraphicsCard_memory, 5*sizeof(int), cudaMemcpyDeviceToHost);
 
    //결과 출력
    for( int i = 0; i < 5; i++)
    {
        printf(" OutputData[%d] : %d\n, i, OutputData[i]);
    }
 
    //그래픽 카드 메모리의 해체
    cudaFree(GraphicsCard_memory);
    */
    return 0;
}
